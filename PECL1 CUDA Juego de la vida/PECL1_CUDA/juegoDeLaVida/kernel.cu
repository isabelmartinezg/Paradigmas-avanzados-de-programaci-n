#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <cstdlib> 
#include <ctime> 
#include <iostream>
#include <string>
#include <stdexcept>
#include "../common/book.h"
using namespace std;

#define viva 'X'
#define muerta ' '

__global__ void kernelMemoriaCompartida(char* Ad, char* Bd, int numColumnas, int numFilas, int TILE_WIDTH)
{
    //Definimos la matriz de memoria compartida
    extern __shared__ char Ads[];

    //Calculamos la posicion, fila y columa de la c�lula
    int fila = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int columna = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int posicion = fila * numColumnas + columna;


    /*Las siguientes comprobaciones se encargan de a�adir una columna y una fila extras a la matriz compartida con el objetivo de poder
    calcular correctamente todos los valores de las c�lulas de la matriz (ya que cada c�lula tiene que acceder a elementos de esa fila
    y columna extras para poder conocer su siguiente valor)*/
    if (blockIdx.x == 0 && threadIdx.x == TILE_WIDTH - 1)
    {
        Ads[posicion + 1] = Ad[posicion + 1];
    }

    if (blockIdx.y == 0 && threadIdx.y == TILE_WIDTH - 1)
    {
        Ads[posicion + numColumnas] = Ad[posicion + numColumnas];
    }

    if (blockIdx.x == 1 && threadIdx.x == 0)
    {
        Ads[posicion - 1] = Ad[posicion - 1];
    }

    if (blockIdx.y == 1 && threadIdx.y == 0)
    {
        Ads[posicion - numColumnas] = Ad[posicion - numColumnas];
    }

    //Sincronizamos todos los hilos para que no haya errores a la hora de comprobar la matriz compartida.
    Ads[posicion] = Ad[posicion];
    __syncthreads();


    /*Las siquientes variables se corresponden a la posici�n de las c�lulas vecinas en la matriz de origen
   para coger el valor correctamente se utiliza el valor de la c�lula correspondiente a cada hilo y, utilizando
   la variable numColumnas (lo que ser�a el Width) se adquieren las posiciones de las vecinas*/
    int diagArribaIzq = posicion - numColumnas - 1;
    int diagArribaDch = posicion - numColumnas + 1;
    int arriba = posicion - numColumnas;
    int abajo = posicion + numColumnas;
    int diagAbajoIzq = posicion + numColumnas - 1;
    int diagAbajoDch = posicion + numColumnas + 1;
    int derecha = posicion + 1;
    int izquierda = posicion - 1;

    /*En lo siquiente guardamos cual es el estado de la c�lula e inicializamos variables que nos servir�n para llevar el
    control del n�mero de vecinas de la c�lula y de cu�ntas de ellas est�n vivas*/
    char estadoCelula = Ads[posicion];
    int nVivas = 0;
    int numVecinas;

    //Esta comprobaci�n se realiza para procesar las posiciones que realmente pertenecen a la matriz real y no pertenecen a hilos que no queremos procesar (se salen de la matriz)
    if (columna < numColumnas || fila < numFilas)
    {
        int* vecinas;

        /*A partir de este momento comienzan todas las comprobaciones para saber cu�les son las c�lulas vecinas a la c�lula en cuesti�n teniendo
        en cuenta la posici�n de la misma. Seg�n donde est� ubicada la c�lula, el n�mero de vecinas var�a, por tanto, en estos condicionales se
        recorren todas las posibilidades para saber con exactitud cu�l es la cantidad de vecinas que tiene la c�lula de inter�s y qu� posici�n
        ocupa cada una de ellas dentro de la matriz.*/
        if (fila == 0) 
        {
            if (columna == 0) 
            {
                vecinas = new int[3]{ derecha,abajo,diagAbajoDch };
                numVecinas = 3;
            }
            else if (columna == numColumnas - 1)
            {
                vecinas = new int[3]{ izquierda,abajo,diagAbajoIzq };
                numVecinas = 3;
            }
            else
            { 
                vecinas = new int[5]{ derecha,abajo,diagAbajoDch,izquierda,diagAbajoIzq };
                numVecinas = 5;
            }

        }
        else if (fila == numFilas - 1) 
        {
            if (columna == 0) 
            {
                vecinas = new  int[3]{ derecha,arriba,diagArribaDch };
                numVecinas = 3;
            }
            else if (columna == numColumnas - 1) 
            {
                vecinas = new int[3]{ izquierda,arriba,diagArribaIzq };
                numVecinas = 3;
            }
            else { 
                vecinas = new int[5]{ derecha,arriba,diagArribaDch,diagArribaIzq,izquierda };
                numVecinas = 5;
            }
        }
        else if (columna == 0) 
        {
            vecinas = new int[5]{ derecha,arriba,diagArribaDch,diagAbajoDch,abajo };
            numVecinas = 5;
        }
        else if (columna == numColumnas - 1) 
        {
            vecinas = new int[5]{ izquierda,arriba,diagArribaIzq,diagAbajoIzq,abajo };
            numVecinas = 5;
        }
        else
        {
            vecinas = new int[8]{ izquierda,arriba,diagArribaIzq,diagAbajoIzq,abajo,diagAbajoDch,diagArribaDch,derecha };

            numVecinas = 8;
        }

        /* Una vez que conocemos el n�mero y la posici�n de las vecinas, recorremos un dichas posiciones en la matriz de origen
       y vamos actualizando el valor de las c�lulas vecinas que est�n vivas*/
        for (int i = 0; i < numVecinas; i++)
        {

            if (Ads[vecinas[i]] == viva)
            {
                nVivas++;
            }
        }

        /* En los siguientes condicionales se actualizar� el estado de la c�lula dependiendo del n�mero de vecinas vivas que tenga*/
        if (estadoCelula == viva)
        {
            if (nVivas == 2 || nVivas == 3)
            {
                estadoCelula = viva;
            }
            else
            {
                estadoCelula = muerta;
            }
        }
        else
        {
            if (nVivas == 3)
            {
                estadoCelula = viva;
            }
        }
        Bd[posicion] = estadoCelula;
    }
}

__global__ void kernelMultiplesBloques(char* Ad, char* Bd, int numColumnas, int numFilas, int TILE_WIDTH)
{

    //Calculamos la posicion, fila y columa de la c�lula
    int fila = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int columna = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int posicion = fila * numColumnas + columna;


    /*Las siquientes variables se corresponden a la posici�n de las c�lulas vecinas en la matriz de origen
   para coger el valor correctamente se utiliza el valor de la c�lula correspondiente a cada hilo y, utilizando
   la variable numColumnas (lo que ser�a el Width) se adquieren las posiciones de las vecinas*/
    int diagArribaIzq = posicion - numColumnas - 1;
    int diagArribaDch = posicion - numColumnas + 1;
    int arriba = posicion - numColumnas;
    int abajo = posicion + numColumnas;
    int diagAbajoIzq = posicion + numColumnas - 1;
    int diagAbajoDch = posicion + numColumnas + 1;
    int derecha = posicion + 1;
    int izquierda = posicion - 1;

    /*En lo siquiente guardamos cual es el estado de la c�lula e inicializamos variables que nos servir�n para llevar el
    control del n�mero de vecinas de la c�lula y de cu�ntas de ellas est�n vivas*/
    char estadoCelula = Ad[posicion];
    int nVivas = 0;
    int numVecinas;

    //Esta comprobaci�n se realiza para que si hay m�s hilos que posiciones de la matriz, no se acceda a posiciones de matriz que no existen
    if (posicion < numFilas * numColumnas)
    {
        int* vecinas;

        /*A partir de este momento comienzan todas las comprobaciones para saber cu�les son las c�lulas vecinas a la c�lula en cuesti�n teniendo
        en cuenta la posici�n de la misma. Seg�n donde est� ubicada la c�lula, el n�mero de vecinas var�a, por tanto, en estos condicionales se
        recorren todas las posibilidades para saber con exactitud cu�l es la cantidad de vecinas que tiene la c�lula de inter�s y qu� posici�n
        ocupa cada una de ellas dentro de la matriz.*/
        if (fila == 0) 
        {
            if (columna == 0) 
            {
                vecinas = new int[3]{ derecha,abajo,diagAbajoDch };
                numVecinas = 3;
            }
            else if (columna == numColumnas - 1) 
            {
                vecinas = new int[3]{ izquierda,abajo,diagAbajoIzq };
                numVecinas = 3;
            }
            else
            { 
                vecinas = new int[5]{ derecha,abajo,diagAbajoDch,izquierda,diagAbajoIzq };
                numVecinas = 5;
            }

        }
        else if (fila == numFilas - 1) 
        {
            if (columna == 0) 
            {
                vecinas = new  int[3]{ derecha,arriba,diagArribaDch };
                numVecinas = 3;
            }
            else if (columna == numColumnas - 1) 
            {
                vecinas = new int[3]{ izquierda,arriba,diagArribaIzq };
                numVecinas = 3;
            }
            else { 
                vecinas = new int[5]{ derecha,arriba,diagArribaDch,diagArribaIzq,izquierda };
                numVecinas = 5;
            }
        }
        else if (columna == 0) 
        {
            vecinas = new int[5]{ derecha,arriba,diagArribaDch,diagAbajoDch,abajo };
            numVecinas = 5;
        }
        else if (columna == numColumnas - 1) 
        {
            vecinas = new int[5]{ izquierda,arriba,diagArribaIzq,diagAbajoIzq,abajo };
            numVecinas = 5;
        }
        else
        {
            vecinas = new int[8]{ izquierda,arriba,diagArribaIzq,diagAbajoIzq,abajo,diagAbajoDch,diagArribaDch,derecha };

            numVecinas = 8;
        }

        /* Una vez que conocemos el n�mero y la posici�n de las vecinas, recorremos un dichas posiciones en la matriz de origen
       y vamos actualizando el valor de las c�lulas vecinas que est�n vivas*/
        for (int i = 0; i < numVecinas; i++)
        {

            if (Ad[vecinas[i]] == viva)
            {
                nVivas++;
            }
        }

        /* En los siguientes condicionales se actualizar� el estado de la c�lula dependiendo del n�mero de vecinas vivas que tenga*/
        if (estadoCelula == viva)
        {
            if (nVivas == 2 || nVivas == 3)
            {
                estadoCelula = viva;
            }
            else
            {
                estadoCelula = muerta;
            }
        }
        else
        {
            if (nVivas == 3)
            {
                estadoCelula = viva;
            }
        }
        Bd[posicion] = estadoCelula;
    }
}

__global__ void kernelUnBloque(char* Ad, char* Bd, int numColumnas, int numFilas)
{
    int posicion = threadIdx.y * numColumnas + threadIdx.x;
    int fila = threadIdx.y;
    int columna = threadIdx.x;

    /*Las siquientes variables se corresponden a la posici�n de las c�lulas vecinas en la matriz de origen
    para coger el valor correctamente se utiliza el valor de la c�lula correspondiente a cada hilo y, utilizando
    la variable numColumnas (lo que ser�a el Width) se adquieren las posiciones de las vecinas*/
    int diagArribaIzq = posicion - numColumnas - 1;
    int diagArribaDch = posicion - numColumnas + 1;
    int arriba = posicion - numColumnas;
    int abajo = posicion + numColumnas;
    int diagAbajoIzq = posicion + numColumnas - 1;
    int diagAbajoDch = posicion + numColumnas + 1;
    int derecha = posicion + 1;
    int izquierda = posicion - 1;

    /*En lo siquiente guardamos cual es el estado de la c�lula e inicializamos variables que nos servir�n para llevar el
    control del n�mero de vecinas de la c�lula y de cu�ntas de ellas est�n vivas*/
    char estadoCelula = Ad[posicion];
    int nVivas = 0;
    int numVecinas;

    //Esta comprobaci�n se realiza para que si hay m�s hilos que posiciones de la matriz, no se acceda a posiciones de matriz que no existen
    if (posicion < numFilas * numColumnas)
    {
        int* vecinas;

        /*A partir de este momento comienzan todas las comprobaciones para saber cu�les son las c�lulas vecinas a la c�lula en cuesti�n teniendo
        en cuenta la posici�n de la misma. Seg�n donde est� ubicada la c�lula, el n�mero de vecinas var�a, por tanto, en estos condicionales se
        recorren todas las posibilidades para saber con exactitud cu�l es la cantidad de vecinas que tiene la c�lula de inter�s y qu� posici�n
        ocupa cada una de ellas dentro de la matriz.*/
        if (fila == 0) 
        {
            if (columna == 0) 
            {
                vecinas = new int[3]{ derecha,abajo,diagAbajoDch };
                numVecinas = 3;
            }
            else if (columna == numColumnas - 1) 
            {
                vecinas = new int[3]{ izquierda,abajo,diagAbajoIzq };
                numVecinas = 3;
            }
            else
            { 
                vecinas = new int[5]{ derecha,abajo,diagAbajoDch,izquierda,diagAbajoIzq };
                numVecinas = 5;
            }

        }
        else if (fila == numFilas - 1) 
        {
            if (columna == 0) 
            {
                vecinas = new  int[3]{ derecha,arriba,diagArribaDch };
                numVecinas = 3;
            }
            else if (columna == numColumnas - 1) 
            {
                vecinas = new int[3]{ izquierda,arriba,diagArribaIzq };
                numVecinas = 3;
            }
            else { 
                vecinas = new int[5]{ derecha,arriba,diagArribaDch,diagArribaIzq,izquierda };
                numVecinas = 5;
            }
        }
        else if (columna == 0) 
        {
            vecinas = new int[5]{ derecha,arriba,diagArribaDch,diagAbajoDch,abajo };
            numVecinas = 5;
        }
        else if (columna == numColumnas - 1) 
        {
            vecinas = new int[5]{ izquierda,arriba,diagArribaIzq,diagAbajoIzq,abajo };
            numVecinas = 5;
        }
        else
        {
            vecinas = new int[8]{ izquierda,arriba,diagArribaIzq,diagAbajoIzq,abajo,diagAbajoDch,diagArribaDch,derecha };

            numVecinas = 8;
        }

        /* Una vez que conocemos el n�mero y la posici�n de las vecinas, recorremos un dichas posiciones en la matriz de origen
        y vamos actualizando el valor de las c�lulas vecinas que est�n vivas*/
        for (int i = 0; i < numVecinas; i++)
        {

            if (Ad[vecinas[i]] == viva)
            {
                nVivas++;
            }
        }


        /* En los siguientes condicionales se actualizar� el estado de la c�lula dependiendo del n�mero de vecinas vivas que tenga*/
        if (estadoCelula == viva)
        {
            if (nVivas == 2 || nVivas == 3)
            {
                estadoCelula = viva;
            }
            else
            {
                estadoCelula = muerta;
            }
        }
        else
        {
            if (nVivas == 3)
            {
                estadoCelula = viva;
            }
        }
        Bd[posicion] = estadoCelula;
        

    }
}


/* La siguiente funci�n tiene como objetivo llenar la matriz de origen con un n�mero especificado de c�lulas vivas repartidas en orden aleatorio.
Para ubicarlas de forma aleatoria se utiliza un n�mero aleatorio.*/
void llenarMatriz(char* A_h, int numFilas, int numColumnas, int Vivas)
{
    srand(time(0));
    int r;
    for (int i = 0; i < numFilas; i++)
    {
        for (int j = 0; j < numColumnas; j++)
        {
            r = rand() % 2;
            if (r == 1 && Vivas > 0 && i > numFilas / 3)
            {
                A_h[j + (i * numColumnas)] = viva;
                Vivas = Vivas - 1;
            }
            else
            {
                A_h[j + (i * numColumnas)] = muerta;
            }
        }
    }
}

int main(int argc, char* argv[])
{
    size_t pos;
    //Las siguientes variables se cogen a partir del comando por consola introducido por el usuario y corresponden al Width y al Heigth
    const int numFilas = stoi(argv[2], &pos);
    const int numColumnas = stoi(argv[3], &pos);
    int numElementos = numFilas * numColumnas;

    size_t size = numElementos * sizeof(char);

    char* A_h;
    char* R_h;
    A_h = new char[numElementos];
    R_h = new char[numElementos];

    char* Ad;
    char* Bd;
    

    /*Cogemos las caracter�sticas del dispositivo*/
    hipDeviceProp_t prop;
    int count;
    int maxThreadsPerBlock;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for (int i = 0; i < count; i++) {


        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        maxThreadsPerBlock = prop.maxThreadsPerBlock;
    }

    //Llenamos la matriz con c�lulas asignando que el n�mero de c�lulas vivas sea un tercio del n�mero total de elementos
    int nVivas = (numFilas * numColumnas) / 3;
    llenarMatriz(A_h, numFilas, numColumnas, nVivas);

    //Asignamos memoria
    hipMalloc((void**)&Ad, size);
    hipMalloc((void**)&Bd, size);

    hipMemcpy(Ad, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, R_h, size, hipMemcpyHostToDevice);

    int modoEjecucion = -1;

    while (modoEjecucion != 1 && modoEjecucion != 2 && modoEjecucion!= 3) {
        cout << "Seleccione el modo de ejecucion del kernel\n 1) Memoria Global 1 Bloque\n 2) Memoria Global Multiples Bloques \n 3) Memoria Compartida ";
        cin >> modoEjecucion;

        //Comprobamos que la configuraci�n del kernel no viole las restricciones del dispositivo
        if (modoEjecucion == 1 && numElementos > maxThreadsPerBlock)
        {
            modoEjecucion = -1;
            printf("No se puede ejecutar en el modo que ha elegido por que las caracteristicas del dispositivo no lo permiten.\n Elija otro modo o reinicie el programa para introducir otra matriz");
        }
    }

    for (int i = 0; i < numFilas; i++)
    {
        for (int j = 0; j < numColumnas; j++)
        {
            printf("%c ", A_h[j + (i * numColumnas)]);
        }
        printf("\n\n");
    }
    if (modoEjecucion == 1)
    {
        dim3 dimGrid(1, 1);
        dim3 dimBlock(numColumnas, numFilas);

        for (int i = 0; i < numFilas; i++)
        {
            //Forma manual
            if (argv[1][1] == 'm')
            {
                int valorUsuario = -1;

                while (valorUsuario != 5)
                {
                    printf("Introduzca 5 para seguir ");
                    scanf("%d", &valorUsuario);
                }
                kernelUnBloque <<< dimGrid, dimBlock >>> (Ad, Bd, numColumnas, numFilas);
                hipMemcpy(R_h, Bd, size, hipMemcpyDeviceToHost);
                hipMemcpy(Ad, Bd, size, hipMemcpyDeviceToDevice);
                for (int i = 0; i < numFilas; i++)
                {
                    for (int j = 0; j < numColumnas; j++)
                    {
                        printf("%c ", R_h[j + (i * numColumnas)]);
                    }
                    printf("\n\n");
                }
            }
            else { //Forma automatica
                Sleep(1000);
                kernelUnBloque <<< dimGrid, dimBlock >>> (Ad, Bd, numColumnas, numFilas);
                hipMemcpy(R_h, Bd, size, hipMemcpyDeviceToHost);
                hipMemcpy(Ad, Bd, size, hipMemcpyDeviceToDevice);
               
                for (int i = 0; i < numFilas; i++)
                {
                    for (int j = 0; j < numColumnas; j++)
                    {
                        printf("%c ", R_h[j + (i * numColumnas)]);
                    }
                    printf("\n\n");
                }
            }
        }
    }else if (modoEjecucion == 2)
    {
        int TILE_WIDTH;
        if (numColumnas % 2 == 0) {
            TILE_WIDTH = numColumnas / 2;
        }
        else
        {
            TILE_WIDTH = (numColumnas / 2) + 1;
        }
        int numBloquesX;
        int numBloquesY;
        if (numColumnas % TILE_WIDTH == 0)
        {
            numBloquesX = numColumnas / TILE_WIDTH;
        }
        else
        {
            numBloquesX = (numColumnas / TILE_WIDTH) + 1;
        }
        if (numFilas % TILE_WIDTH == 0)
        {
            numBloquesY = numFilas / TILE_WIDTH;
        }
        else
        {
            numBloquesY = (numFilas / TILE_WIDTH) + 1;
        }
        dim3 dimGrid(numBloquesX, numBloquesY);
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

        //Comprobamos que la configuraci�n del kernel no viole las restricciones del dispositivo
        if ((TILE_WIDTH * TILE_WIDTH)> maxThreadsPerBlock)
        {
            printf("Error, pruebe con otra matriz, la matriz introducida no est� admitida por las caracter�sticas del dispositivo.");
        }
        else
        {
            for (int i = 0; i < numFilas; i++)
            {
                //Forma manual
                if (argv[1][1] == 'm')
                {
                    int valorUsuario = -1;

                    while (valorUsuario != 5)
                    {
                        printf("Introduzca 5 para seguir ");
                        scanf("%d", &valorUsuario);
                    }
                    kernelMultiplesBloques << < dimGrid, dimBlock >> > (Ad, Bd, numColumnas, numFilas, TILE_WIDTH);
                    hipMemcpy(R_h, Bd, size, hipMemcpyDeviceToHost);
                    hipMemcpy(Ad, Bd, size, hipMemcpyDeviceToDevice);
                    for (int i = 0; i < numFilas; i++)
                    {
                        for (int j = 0; j < numColumnas; j++)
                        {
                            printf("%c ", R_h[j + (i * numColumnas)]);
                        }
                        printf("\n\n");
                    }
                }
                else { //Forma automatica
                    Sleep(1000);
                    kernelMultiplesBloques << < dimGrid, dimBlock >> > (Ad, Bd, numColumnas, numFilas, TILE_WIDTH);
                    hipMemcpy(R_h, Bd, size, hipMemcpyDeviceToHost);
                    hipMemcpy(Ad, Bd, size, hipMemcpyDeviceToDevice);
                    
                    for (int i = 0; i < numFilas; i++)
                    {
                        for (int j = 0; j < numColumnas; j++)
                        {
                            printf("%c ", R_h[j + (i * numColumnas)]);
                        }
                        printf("\n\n");
                    }
                }
            }
        }
        
    }
    else if (modoEjecucion == 3)
    {
   
        int TILE_WIDTH;
        if (numColumnas % 2 == 0) {
            TILE_WIDTH = numColumnas / 2;
        }
        else
        {
            TILE_WIDTH = (numColumnas / 2) + 1;
        }
        int numBloquesX;
        int numBloquesY;
        if (numColumnas % TILE_WIDTH == 0)
        {
            numBloquesX = numColumnas / TILE_WIDTH;
        }
        else
        {
            numBloquesX = (numColumnas / TILE_WIDTH) + 1;
        }
        if (numFilas % TILE_WIDTH == 0)
        {
            numBloquesY = numFilas / TILE_WIDTH;
        }
        else
        {
            numBloquesY = (numFilas / TILE_WIDTH) + 1;
        }
        dim3 dimGrid(numBloquesX, numBloquesY);
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

       
        for (int i = 0; i < numFilas; i++)
        {
            int numBloquesX;
            int numBloquesY;

            if (numColumnas % TILE_WIDTH == 0)
            {
                numBloquesX = numColumnas / TILE_WIDTH;
            }
            else
            {
                numBloquesX = (numColumnas / TILE_WIDTH) + 1;
            }
            if (numFilas % TILE_WIDTH == 0)
            {
                numBloquesY = numFilas / TILE_WIDTH;
            }
            else
            {
                numBloquesY = (numFilas / TILE_WIDTH) + 1;
            }

            dim3 dimGrid(numBloquesX, numBloquesY);
            dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
            if (argv[1][1] == 'm')
            {
                int valorUsuario = -1;

                while (valorUsuario != 5)
                {
                    printf("Introduzca 5 para seguir ");
                    scanf("%d", &valorUsuario);
                }
                kernelMemoriaCompartida << < dimGrid, dimBlock, ((TILE_WIDTH * TILE_WIDTH) + 1) * sizeof(char*) >> > (Ad, Bd, numColumnas, numFilas, TILE_WIDTH);
                hipMemcpy(R_h, Bd, size, hipMemcpyDeviceToHost);
                hipMemcpy(Ad, Bd, size, hipMemcpyDeviceToDevice);
                for (int i = 0; i < numFilas; i++)
                {
                    for (int j = 0; j < numColumnas; j++)
                    {
                        printf("%c ", R_h[j + (i * numColumnas)]);
                    }
                    printf("\n\n");
                }
            }
            else { //Forma automatica
                Sleep(1000);
                kernelMemoriaCompartida << < dimGrid, dimBlock, ((TILE_WIDTH * TILE_WIDTH) + 1) * sizeof(char*) >> > (Ad, Bd, numColumnas, numFilas, TILE_WIDTH);
                hipMemcpy(R_h, Bd, size, hipMemcpyDeviceToHost);
                hipMemcpy(Ad, Bd, size, hipMemcpyDeviceToDevice);
                for (int i = 0; i < numFilas; i++)
                {
                    for (int j = 0; j < numColumnas; j++)
                    {
                        printf("%c ", R_h[j + (i * numColumnas)]);
                    }
                    printf("\n\n");
                }
            }

        }   
    }
    //Liberamos espacio
    hipFree(Bd);
    hipFree(Ad);

    return 0;
}
